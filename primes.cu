#include <iostream>
#include <stdint.h>
#include <string.h>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <ctime>
#include <hiprand/hiprand_kernel.h>

#include "util.h"
#include "big_integer.h"
#include "primes.h"

bigi cluster_offsets[6] = {0, 4, 2, 4, 2, 4};

__device__
bool is_even(bigi n) {
  bigi one = bigi(1);
  bigi zero = bigi(0);
  bigi x = n & one;
  return x == zero;
}

__device__
bigi mod_exp(bigi base, bigi exponent, bigi mod) {
  bigi result = 1;
  bigi base_acc = base;
  bigi exp_acc = exponent;
  bigi one = bigi(1);
  while (exp_acc > bigi(0)) {
    bigi x = exp_acc & one;
    if (x == one) {
      bigi temp = result * base_acc;
      result = temp;

      result = result % mod;
    }
    base_acc = base_acc * base_acc;
    base_acc = base_acc % mod;
    exp_acc >>= one;
  }
  return result;
}

__device__
bool is_prime_fermat(bigi n, int k=1000) {
  bigi rem, a;

  hiprandState state;
  hiprand_init((unsigned int)n, 0, 0, &state);

  if (n == bigi(2)) {
    return true;
  } else if (is_even(n)) {
    return false;
  }

  for (int i = 0; i < k; i++) {
    bigi nmo = n - bigi(1);
    a = bigi(hiprand(&state)) % nmo;
    a += 1; // 1 <= a < n
    rem = mod_exp(a, nmo, n);
    if (rem != bigi(1))
      return false;
  }

  return true;
}
struct is_valid_pow {

  __device__
  bool operator()(const bigi candidate) {
    bigi cluster_offsets[6] = {0, 4, 2, 4, 2, 4};
    bigi val = candidate;
    for (int i = 0; i < 6; i++) {
      val += cluster_offsets[i];
      if (!is_prime_fermat(val))
        return false;
    }
    return true;
  }

};

void sieve(uint32_t** prime_test_table, uint32_t* prime_test_size,
    uint32_t prime_test_limit) {
  /* Source: http://git.io/-m1Ypw
   * We use uint8 instead of bool because otherwise our array would be too
   * large to index into. */
  *prime_test_table = (uint32_t*)malloc(sizeof(uint32_t)
      *(prime_test_limit/4+10));
  if (prime_test_table == NULL) {
    perror("could not allocate prime test table");
    exit(-1);
  }
  *prime_test_size = 0;

  uint8_t* vfComposite = (uint8_t*)malloc(sizeof(uint8_t)
      *(prime_test_limit+7)/8);
  if (vfComposite == NULL) {
    perror("could not allocate vfComposite table");
    exit(-1);
  }
  memset(vfComposite, 0x00, sizeof(uint8_t)*(prime_test_limit+7)/8);
  for (unsigned int nFactor = 2; nFactor * nFactor < prime_test_limit;
      nFactor++) {
    if(vfComposite[nFactor>>3] & (1<<(nFactor&7)))
      continue;
    for (unsigned int nComposite = nFactor * nFactor;
        nComposite < prime_test_limit; nComposite += nFactor)
      vfComposite[nComposite>>3] |= 1<<(nComposite&7);
  }
  for (unsigned int n = 2; n < prime_test_limit; n++) {
    if ((vfComposite[n>>3] & (1<<(n&7))) == 0) {
      (*prime_test_table)[*prime_test_size] = n;
      (*prime_test_size)++;
    }
  }
  *prime_test_table = (uint32_t*)realloc(*prime_test_table,
    sizeof(uint32_t)*(*prime_test_size));
  free(vfComposite);
}

bool candidate_killed_by(bigi candidate, bigi prime) {
  bigi p = candidate;
  for(int i = 0; i < 6; i++) {
    p += cluster_offsets[i];
    if (p % prime == bigi(0))
      return true;
  }
  return false;
}


thrust::host_vector<bigi> add_next_prime(thrust::host_vector<bigi> offsets,
    bigi max_val, bigi prime, bigi primorial) {
  bigi base = 0;
  bigi counter = 0;

  thrust::host_vector<bigi> new_offsets;

  while (counter < prime) {
    if (base > max_val)
      break;
    for (thrust::host_vector<bigi>::iterator o = offsets.begin();
        o != offsets.end(); o++) {
      bigi val = base + *o;
      if (val > max_val)
        break;
      if (!candidate_killed_by(val, prime))
        new_offsets.push_back(val);
    }
    base + primorial;
    counter += 1;
  }

  return new_offsets;
}

std::vector<bigi> generate_prime_clusters(bigi max_val, uint32_t max_sieve,
    bool verbose) {
  uint32_t primorial_start = 7;

  bigi primorial = 210;
  bigi big97 = 97;

  thrust::host_vector<bigi> offsets;
  offsets.push_back(big97);

  uint32_t *prime_test_table;
  uint32_t prime_test_size;
  DEBUG("Starting sieve")
  sieve(&prime_test_table, &prime_test_size, max_sieve);
  DEBUG("Finished sieve")

  DEBUG("Starting adding primes")

  for (uint32_t i = 0; i < prime_test_size; i++) {
    if (prime_test_table[i] <= primorial_start)
      continue;
    bigi prime = prime_test_table[i];
    offsets = add_next_prime(offsets, max_val, prime, primorial);
    primorial = primorial * prime;
    DEBUG(prime_test_table[i])
  }
  DEBUG("Finished adding primes")
  DEBUG("Checking if PoWs")

  thrust::device_vector<bigi> candidates = offsets;
  thrust::device_vector<bigi> clusters;

  thrust::copy_if(candidates.begin(), candidates.end(), clusters.begin(), is_valid_pow());

  thrust::host_vector<bigi> result = clusters;
  std::vector<bigi> primes(result.begin(), result.end());

  return primes;
}
