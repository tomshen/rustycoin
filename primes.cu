#include <iostream>
#include <stdint.h>
#include <string.h>
#include <vector>

#include <gmpxx.h>
#include <ctime>
#include <hiprand/hiprand_kernel.h>

#include "primes.h"
#include "util.h"
#include "big_integer.h"

#define bigi thrust::big_integer<300>
#define bigi_t thrust::big_integer<300>

static bigi cluster_offsets[6] = {0, 4, 2, 4, 2, 4};

__device__
bool is_even(bigi_t n) {
  bigi one = bigi(1);
  bigi zero = bigi(0);
  bigi x = n & one;
  return x == zero;
}

__device__
bigi mod_exp(bigi base, bigi exponent, bigi mod) {
  bigi result = 1;
  bigi base_acc = base;
  bigi exp_acc = exponent;
  bigi one = bigi(1);
  while (exp_acc > bigi(0)) {
    bigi x = exp_acc & one;
    if (x == one) {
      bigi temp = result * base_acc;
      result = temp;

      result = result % mod;
    }
    base_acc = base_acc * base_acc;
    base_acc = base_acc % mod;
    exp_acc >>= one;
  }
  return result;
}

__device__
bool is_prime_fermat(bigi n, int k=1000) {
  bigi rem, a;

  hiprandState state;
  hiprand_init((unsigned int)n, 0, 0, &state);

  if (n == bigi(2)) {
    return true;
  } else if (is_even(n)) {
    return false;
  }

  for (int i = 0; i < k; i++) {
    bigi nmo = n - bigi(1);
    a = bigi(hiprand(&state)) % nmo;
    a += 1; // 1 <= a < n
    rem = mod_exp(a, nmo, n);
    if (rem != bigi(1))
      return false;
  }

  return true;
}

__device__
bool is_valid_pow(bigi candidate) {
  bigi cluster_offsets[6] = {0, 4, 2, 4, 2, 4};
  bigi val = candidate;
  for (int i = 0; i < 6; i++) {
    val += cluster_offsets[i];
    if (!is_prime_fermat(val))
      return false;
  }
  return true;
}

void sieve(uint32_t** prime_test_table, uint32_t* prime_test_size,
    uint32_t prime_test_limit) {
  /* Source: http://git.io/-m1Ypw
   * We use uint8 instead of bool because otherwise our array would be too
   * large to index into. */
  *prime_test_table = (uint32_t*)malloc(sizeof(uint32_t)
      *(prime_test_limit/4+10));
  if (prime_test_table == NULL) {
    perror("could not allocate prime test table");
    exit(-1);
  }
  *prime_test_size = 0;

  uint8_t* vfComposite = (uint8_t*)malloc(sizeof(uint8_t)
      *(prime_test_limit+7)/8);
  if (vfComposite == NULL) {
    perror("could not allocate vfComposite table");
    exit(-1);
  }
  memset(vfComposite, 0x00, sizeof(uint8_t)*(prime_test_limit+7)/8);
  for (unsigned int nFactor = 2; nFactor * nFactor < prime_test_limit;
      nFactor++) {
    if(vfComposite[nFactor>>3] & (1<<(nFactor&7)))
      continue;
    for (unsigned int nComposite = nFactor * nFactor;
        nComposite < prime_test_limit; nComposite += nFactor)
      vfComposite[nComposite>>3] |= 1<<(nComposite&7);
  }
  for (unsigned int n = 2; n < prime_test_limit; n++) {
    if ((vfComposite[n>>3] & (1<<(n&7))) == 0) {
      (*prime_test_table)[*prime_test_size] = n;
      (*prime_test_size)++;
    }
  }
  *prime_test_table = (uint32_t*)realloc(*prime_test_table,
    sizeof(uint32_t)*(*prime_test_size));
  free(vfComposite);
}
/*
bool candidate_killed_by(mpz_class candidate, mpz_class prime) {
  mpz_class p = candidate;
  for(int i = 0; i < 6; i++) {
    p += cluster_offsets[i];
    if (p % prime == 0)
      return true;
  }
  return false;
}


std::vector<mpz_class> add_next_prime(std::vector<mpz_class> offsets,
    mpz_class max_val, mpz_class prime, mpz_class primorial) {
  mpz_class base = 0;
  mpz_class counter = 0;

  std::vector<mpz_class> new_offsets;

  while (counter < prime) {
    if (base > max_val)
      break;
    for (std::vector<mpz_class>::iterator o = offsets.begin();
        o != offsets.end(); o++) {
      mpz_class val = base + *o;
      if (val > max_val)
        break;
      if (!candidate_killed_by(val, prime))
        new_offsets.push_back(val);
    }
    base + primorial;
    counter += 1;
  }

  return new_offsets;
}

uint32_t generate_prime_clusters(mpz_class max_val, uint32_t max_sieve,
    bool verbose) {
  uint32_t primorial_start = 7;

  mpz_class primorial = 210;
  mpz_class big97 = 97;

  std::vector<mpz_class> offsets;
  offsets.push_back(big97);

  uint32_t *prime_test_table;
  uint32_t prime_test_size;
  sieve(&prime_test_table, &prime_test_size, max_sieve);

  for (uint32_t i = primorial_start+1; i < prime_test_size; i++) {
    mpz_class prime = i;
    offsets = add_next_prime(offsets, max_val, prime, primorial);
    primorial *= i;
  }
  uint32_t count = 0;
  for (std::vector<mpz_class>::iterator o = offsets.begin();
      o != offsets.end(); o++) {
    if (is_valid_pow(*o)) {
      count++;
      if (verbose)
        std::cout << "Found valid POW " << *o << std::endl;
    }
  }
  
  thrust::filter(
  return count;
}
*/
